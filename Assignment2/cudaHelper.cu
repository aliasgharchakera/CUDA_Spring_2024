#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>
#include <hip/hip_runtime.h>

inline hipError_t checkCudaErr(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
    }
    return err;
}

__global__ void cudaMatrixMulKernel(float *M, float *N, float *P, int rows,
                                    int cols, int common)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        float Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < common; ++k)
        {
            Pvalue += M[k * rows + row] * N[col * common + k];
        }
        P[col * rows + row] = Pvalue;
    }
}

__global__ void cudaMatrixScalarMulKernel(float *M, float N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] * N;
    }
}

Eigen::MatrixXf cudaMatrixMul(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = N.cols();
    if (M.cols() != N.rows())
    {
        std::cout << M.rows() << "," << M.cols() << std::endl;
        std::cout << N.rows() << "," << N.cols() << std::endl;
        std::cout << "Matrix dimensions are not compatible for multiplication" << std::endl;
        return Eigen::MatrixXf::Zero(1, 1);
    }
    int common = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * common * sizeof(float);
    int size_N = common * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);

    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_N);
    hipMalloc((void **)&d_P, size_P);
    checkCudaErr(hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice), "Memcpy M");
    checkCudaErr(hipMemcpy(d_N, N.data(), size_N, hipMemcpyHostToDevice), "Memcpy N");

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    cudaMatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols, common);
    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU Error");

    Eigen::MatrixXf P(rows, cols);
    checkCudaErr(hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost), "Memcpy P");
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}

Eigen::MatrixXf cudaMatrixScalarMul(const Eigen::MatrixXf &M, float N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_P, size_P);
    checkCudaErr(hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice), "Memcpy M");

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    cudaMatrixScalarMulKernel<<<dimGrid, dimBlock>>>(d_M, N, d_P, rows, cols);
    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU Error");

    Eigen::MatrixXf P(rows, cols);
    checkCudaErr(hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost), "Memcpy P");
    hipFree(d_M);
    hipFree(d_P);

    return P;
}

__global__ void cudaMatrixAddKernel(float *M, float *N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] + N[col * rows + row];
    }
}

Eigen::MatrixXf cudaMatrixAdd(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_M);
    hipMalloc((void **)&d_P, size_P);
    checkCudaErr(hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice), "Memcpy M");
    checkCudaErr(hipMemcpy(d_N, N.data(), size_M, hipMemcpyHostToDevice), "Memcpy N");

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    cudaMatrixAddKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols);
    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU Error");

    Eigen::MatrixXf P(rows, cols);
    checkCudaErr(hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost), "Memcpy P");
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}

__global__ void cudaMatrixSubKernel(float *M, float *N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] - N[col * rows + row];
    }
}

Eigen::MatrixXf cudaMatrixSub(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_M);
    hipMalloc((void **)&d_P, size_P);
    checkCudaErr(hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice), "Memcpy M");
    checkCudaErr(hipMemcpy(d_N, N.data(), size_M, hipMemcpyHostToDevice), "Memcpy N");

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    cudaMatrixSubKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols);
    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU Error");

    Eigen::MatrixXf P(rows, cols);
    checkCudaErr(hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost), "Memcpy P");
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}